#include "hip/hip_runtime.h"
#include"../inc/matrix_operations.cuh"

#define STREAMS 8

__constant__ size_t cN;

// // //

__global__ void transpose(float* in, float* out, unsigned int nx, unsigned int ny, unsigned int padding)
{
    extern __shared__ float tile[];

    unsigned int in_idx, out_idx;

    unsigned int ix = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;

    in_idx = iy*nx + ix;

    unsigned int block_idx, block_row, block_col;
    block_idx = threadIdx.y * blockDim.x + threadIdx.x;
    block_row = block_idx/blockDim.y;
    block_col = block_idx%blockDim.y;    

    unsigned int ox = blockIdx.y*blockDim.y + block_col;
    unsigned int oy = blockIdx.x*blockDim.x*2 + block_row;

    out_idx =  oy* ny + ox;

    if(ix + blockDim.x < nx && iy < ny)
    {
        unsigned int row_idx = threadIdx.y * (blockDim.x *2 + padding) + threadIdx.x;
        tile[row_idx] = in[in_idx];
        tile[row_idx+blockDim.x] = in[in_idx + blockDim.x];

        __syncthreads();

        unsigned int col_idx = block_col * (blockDim.x *2 + padding) + block_row;
        out[out_idx] = tile[col_idx];
        out[out_idx+ny*blockDim.x] = tile[col_idx + blockDim.x];
    }
}

//

void add(float* dest, float* A, float *B, size_t r, size_t c, dim3 blockSize)
{
    size_t N = r*c;
    dim3 gridSize = {(N+blockSize.x-1)/blockSize.x, (N+blockSize.y-1)/blockSize.y, 1};
    hipStream_t* streams = (hipStream_t*) malloc(sizeof(hipStream_t)*STREAMS);
    for(int i = 0; i < STREAMS; i++)
        hipStreamCreate(&streams[i]);
    size_t chunk = (N + STREAMS - 1)/STREAMS;

    float* A_D, *B_D, *dest_D;
    hipMalloc(&A_D, sizeof(float)*N);
    hipMalloc(&B_D, sizeof(float)*N);
    hipMalloc(&dest_D, sizeof(float)*N);
    hipMemcpyToSymbol(HIP_SYMBOL(cN), &N, sizeof(size_t));
    for(int i = 0; i < STREAMS; i++ )
    {
        size_t offset = i*chunk;
        hipMemcpyAsync(&A_D[offset], &A[offset], sizeof(float)*chunk, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&B_D[offset], &B[offset], sizeof(float)*chunk, hipMemcpyHostToDevice, streams[i]);
        add<<<blockSize, gridSize,0,streams[i]>>>(dest_D, A_D, B_D);
        hipMemcpyAsync(&dest[offset], &dest_D[offset], sizeof(float)*chunk,hipMemcpyDeviceToHost, streams[i]);
    }
    for(int i = 0; i < STREAMS; i++)
        hipStreamDestroy(streams[i]);
    hipFree(A_D);
    hipFree(B_D);
    hipFree(dest_D);
}

__global__ void add(float* dest, float* A, float* B)
{
    size_t idx = blockIdx.y * gridDim.x + blockIdx.x + threadIdx.y * blockDim.x + threadIdx.x;
    if(idx < cN) dest[idx] = A[idx] + B[idx];
}